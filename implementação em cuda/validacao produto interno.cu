// Validando a transposição, produto interno e 
// aplicação do soft max em multi head


#include <hip/hip_runtime.h>
#include "stdio.h"
#include "cmath"

// constantes no modelo gpt-3, para referência
// const int d_model = 12288;
// const int n_heads = 96;
// const int C = 2048; // TamaHo do context window
// const int vocabulary_size = 50257;
// const int D = d_model / H;
// const int thread_size = 32;

// constantes meu modelo
const int d_model = 4;
const int n_heads = 2;
const int C = 2; // TamaHo do context window
const int D = d_model / n_heads; // = 2
const int block_dim_x = 8, block_dim_y = 8, block_dim_z = 16;
const double sqrtD = sqrtl(D); 

// Teto da divisão de a por b
int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

// Pega um número aleatório entre -1 e 1
double rand_double() {
  double min = -1, max = 1;
  double range = max - min;
  return (double) rand() / (RAND_MAX / range);
}

// Aplica uma camada linear A de forma (k, m) a uma matriz B de forma (n, m)
// para obter uma matriz resultante C de forma (n, k)
__global__ void transformacao_linear(double *A, double *B, double *C, int n, int m, int k, int n_heads) {
  int h = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;
  int j = threadIdx.z + blockDim.z * blockIdx.z;
  
  // i é o índice do vetor que vamos aplicar a camada agora
  // j é qual dimensão da camada estamos agora para aplicar a transformação
  if (h < n_heads and i < n and j < k) {
    double soma = 0;
    for (int idx = 0; idx < m; ++idx) {
      soma += A[(h * k * m) + (j * m) + idx] * B[(i * m) + idx];
    }
    C[(h * n * k) + (i * k) + j] = soma;
  }
}

// Transpõe uma matriz A de forma (n, m) e coloca o resultado 
// em B, que tem forma (m, n)
__global__ void transpor(double *A, double *B, int n, int m, int n_heads) {
  int h = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;
  int j = threadIdx.z + blockDim.z * blockIdx.z;

  if (h < n_heads and i < n and j < m) {
    B[(h * m * n) + (j * n) + i] = A[(h * n * m) + (i * m) + j];
  }
}

void transpor_cpu(double *K, double *K_transposto, int C, int D, int n_heads) {
  for (int h = 0; h < n_heads; ++h) {
    for (int i = 0; i < C; ++i) {
      for (int j = 0; j < D; ++j) {
        K_transposto[(h * D * C) + (j * C) + i] = K[(h * C * D) + (i * D) + j];
      }
    }
  }
}

// Multiplica uma matriz A de forma (n, k) com uma matriz B de forma (k, m) 
// e coloca o resultado na matriz C, que tem forma (n, m)
// Nesta função, aplicamos o conceito de masked self attention,
// para que neHum token obteHa informações sobre tokens em posições à frente
// e dividimos todo valor por D por questões de normalização de valores
__global__ void primeira_multiplicacao(double *A, double *B, double *C, int n, int m, int k, double sqrtD, int n_heads) {
  int h = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  int i = threadIdx.z + blockDim.z * blockIdx.z;

  if (h < n_heads and j < m and i < n) {
    // No caso de masked attention, só calculamos o produto interno 
    // Q_i * K_j quando j <= i, para prevenir "spoilers" pro modelo
    if (j <= i) {
      double soma = 0;
      for (int idx = 0; idx < k; ++idx) {
        soma += A[(h * n * k) + (i * k) + idx] * B[(h * k * m) + (idx * m) + j];
      }
      C[(h * n * m) + (i * m) + j] = soma / sqrtD; // A divisão normaliza o resultado 
    } else {
      C[(h * n * m) + (i * m) + j] = -INFINITY; 
    }
  }
}

void produto_interno_escalado(double *Q, double *K_transposto, double *host_H, int C, int D, int n_heads) {
  for (int h = 0; h < n_heads; ++h) {
    for (int j = 0; j < C; ++j) {
      for (int i = 0; i < C; ++i) {
        if (j <= i) {
          double soma = 0;
          for (int idx = 0; idx < D; ++idx) {
            soma += Q[(h * C * D) + (i * D) + idx] * K_transposto[(h * D * C) + (idx * C) + j];
          }
          host_H[(h * C * C) + (i * C) + j] = soma / sqrtD;
        } else {
          host_H[(h * C * C) + (i * C) + j] = -INFINITY;
        }
      }
    }
  }
}

// Pega uma matriz A de forma (n, m) e aplica a função softmax
// em cada uma de suas liHas
__global__ void softmax(double *A, int n, int m, int n_heads){
  int h = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  if (h < n_heads and i < n) {
    double soma = 0;
    for (int idx = 0; idx < m; ++idx) {
      if (A[(h * n * m) + (i * m) + idx] != -INFINITY) { // caso contrário, ele não vai conseguir calcular a exponencial 
        soma += exp(A[(h * n * m) + (i * m) + idx]);
      }
    }
    for (int idx = 0; idx < m; ++idx) {
      if (A[(h * n * m) + (i * m) + idx] != -INFINITY) {
        A[(h * n * m) + (i * m) + idx] = exp(A[(h * n * m) + (i * m) + idx]) / soma;
      } else {
        A[(h * n * m) + (i * m) + idx] = 0;
      }
    }
  }
}

void softmax_cpu(double *host_H, int C, int n_heads) {
  for (int h = 0; h < n_heads; ++h) {
    for (int i = 0; i < C; ++i) {
      double soma = 0;
      for (int j = 0; j < C; ++j) {
        if (host_H[(h * C * C) + (i * C) + j] != -INFINITY) {
          soma += exp(host_H[(h * C * C) + (i * C) + j]);
        }
      }
      for (int j = 0; j < C; ++j) {
        if (host_H[(h * C * C) + (i * C) + j] == -INFINITY) {
          host_H[(h * C * C) + (i * C) + j] = 0;
        } else {
          host_H[(h * C * C) + (i * C) + j] = exp(host_H[(h * C * C) + (i * C) + j]) / soma;
        }
      }
    }
  }
}

struct MultiHeadAttention{
  double *host_W_V, *host_W_Q, *host_W_K;
  double *device_W_V, *device_W_Q, *device_W_K, 
         *device_V, *device_Q, *device_K,
         *device_K_transposto;
  double *device_E, *device_H;

  MultiHeadAttention(){
    // As seguintes são camadas lineares que transformam a
    // matriz de embedding da forma (C, d_model) para (C, D)
    host_W_V = (double *) malloc(n_heads * D * d_model * sizeof(double));
    host_W_Q = (double *) malloc(n_heads * D * d_model * sizeof(double));
    host_W_K = (double *) malloc(n_heads * D * d_model * sizeof(double));

    // Define inicialmente valores aleatórios para cada 
    // valor em W_V, W_Q, e W_K
    for (int i = 0; i < n_heads * D * d_model; ++i) {
      host_W_V[i] = rand_double();
      host_W_Q[i] = rand_double();
      host_W_K[i] = rand_double();
    }

    // Colocando essas matrizes na GPU
    hipMalloc(&device_W_V, n_heads * D * d_model * sizeof(double));
    hipMalloc(&device_W_Q, n_heads * D * d_model * sizeof(double));
    hipMalloc(&device_W_K, n_heads * D * d_model * sizeof(double));
    hipMemcpy(device_W_V, host_W_V, n_heads * D * d_model * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_W_Q, host_W_Q, n_heads * D * d_model * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_W_K, host_W_K, n_heads * D * d_model * sizeof(double), hipMemcpyHostToDevice);

    // Alocando espaço para as matrizes 
    // V, Q, K. Estas serão as matrizes que
    // resultam das projeções da matriz de 
    // embedding nas camadas lineares
    hipMalloc(&device_V, n_heads * C * D * sizeof(double));
    hipMalloc(&device_Q, n_heads * C * D * sizeof(double));
    hipMalloc(&device_K, n_heads * C * D * sizeof(double));

    // Aloca espaço para copiarmos a matriz de embedding para a GPU
    // (não precisa multiplicar por H porque é único através das heads)
    hipMalloc(&device_E, C * d_model * sizeof(double)); 

    // Prepara a matriz transposta de K na GPU
    hipMalloc(&device_K_transposto, n_heads * D * C * sizeof(double));

    // Matriz auxiliar para fazer a multiplicação entre Q e K^T
    hipMalloc(&device_H, n_heads * C * C * sizeof(double));
  }

  void pass_embedding(double *E){
    // E é o embedding do input. Ele tem forma (C, d_model),
    // onde C é o tamanHo da janela de contexto e d_model é 
    // a dimensão do modelo. Primeiro, temos que fazer uma 
    // transformação linear para transformar E de (C, d_model)
    // para (C, D), para termos exatamente C * d_model valores 
    // nos embeddings entre todas as heads. 

    // Copia E para a GPU
    hipMemcpy(device_E, E, C * d_model * sizeof(double), hipMemcpyHostToDevice);

    // Faz as transformações lineares paralelamente
    dim3 grid_dim_tl(ceil_div(n_heads, block_dim_x), ceil_div(C, block_dim_y), ceil_div(D, block_dim_z));
    dim3 block_dim_tl(block_dim_x, block_dim_y, block_dim_z);
    transformacao_linear<<<grid_dim_tl, block_dim_tl>>>(device_W_V, device_E, device_V, C, d_model, D, n_heads);
    transformacao_linear<<<grid_dim_tl, block_dim_tl>>>(device_W_Q, device_E, device_Q, C, d_model, D, n_heads);
    transformacao_linear<<<grid_dim_tl, block_dim_tl>>>(device_W_K, device_E, device_K, C, d_model, D, n_heads);
    hipDeviceSynchronize();

    // copiando resultados na CPU:
    double *host_Q, *host_K;
    host_Q = (double *) malloc(n_heads * C * D * sizeof(double));
    host_K = (double *) malloc(n_heads * C * D * sizeof(double));
    hipMemcpy(host_Q, device_Q, n_heads * C * D * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host_K, device_K, n_heads * C * D * sizeof(double), hipMemcpyDeviceToHost);

    // Vamos transpor a matriz K para podermos fazer a multiplicação Q(K^T) 
    dim3 grid_dim_t = grid_dim_tl;
    dim3 block_dim_t = block_dim_tl;
    transpor<<<grid_dim_t, block_dim_t>>>(device_K, device_K_transposto, C, D, n_heads);
    hipDeviceSynchronize();

    // --- Olhando transposição ---
    // Fazendo a transposição na CPU
    double *host_K_transposto;
    host_K_transposto = (double *) malloc(n_heads * D * C * sizeof(double));
    transpor_cpu(host_K, host_K_transposto, C, D, n_heads);

    // Cópia da transposição da GPU para a CPU:
    double *device_copy_K_transposto = (double *) malloc(n_heads * D * C * sizeof(double));
    hipMemcpy(device_copy_K_transposto, device_K_transposto, n_heads * D * C * sizeof(double), hipMemcpyDeviceToHost);

    printf("Matrizes K transpostas GPU:\n");
    for (int h = 0; h < n_heads; ++h) {
      printf("head: %d\n", h);
      for (int i = 0; i < D; ++i) {
        for (int j = 0; j < C; ++j){
          printf("%lf ", device_copy_K_transposto[(h * D * C) + (i * C) + j]);
        }
        printf("\n");
      }
    }
    printf("matrizes K transpostas (CPU):\n");
    for (int h = 0; h < n_heads; ++h) {
      printf("head: %d\n", h);
      for (int i = 0; i < D; ++i) {
        for (int j = 0; j < C; ++j) {
          printf("%lf ", host_K_transposto[(h * D * C) + (i * C) + j]);
        }
        printf("\n");
      }
    }
    // --- --- 
    
    // Fazemos a multiplicação, gerando uma matriz H de forma (C, C):
    dim3 grid_dim_pm(ceil_div(n_heads, block_dim_x), ceil_div(C, block_dim_y), ceil_div(C, block_dim_z));
    dim3 block_dim_pm = block_dim_tl;
    primeira_multiplicacao<<<grid_dim_pm, block_dim_pm>>>(device_Q, device_K_transposto, device_H, C, C, D, sqrtD, n_heads);
    hipDeviceSynchronize();

    // --- olhando produto interno ---
    // Fazendo esse produto na cpu
    double *host_H = (double *) malloc(n_heads * C * C * sizeof(double));
    produto_interno_escalado(host_Q, host_K_transposto, host_H, C, D, n_heads);

    double *device_copy_H_presoftmax = (double *) malloc(n_heads * C * C * sizeof(double));
    hipMemcpy(device_copy_H_presoftmax, device_H, n_heads * C * C * sizeof(double), hipMemcpyDeviceToHost);
    printf("Matrizes H presoftmax GPU:\n");
    for (int h = 0; h < n_heads; ++h) {
      printf("head: %d\n", h);
      for (int i = 0; i < C; ++i) {
        for (int j = 0; j < C; ++j){
          printf("%lf ", device_copy_H_presoftmax[(h * C * C) + (i * C) + j]);
        }
        printf("\n");
      }
    }
    
    printf("Matrizes H presoftmax CPU:\n");
    for (int h = 0; h < n_heads; ++h) {
      printf("head: %d\n", h);
      for (int i = 0; i < C; ++i) {
        for (int j = 0; j < C; ++j){
          printf("%lf ", host_H[(h * C * C) + (i * C) + j]);
        }
        printf("\n");
      }
    }
    // --- ---
    
    // Aplicamos o softmax na matriz resultante da última multiplicação:
    dim3 grid_dim_s(ceil_div(n_heads, block_dim_x), ceil_div(C, block_dim_y));
    dim3 block_dim_s(block_dim_x, block_dim_y);
    softmax<<<grid_dim_s, block_dim_s>>>(device_H, C, C, n_heads);
    hipDeviceSynchronize();

    // --- Olhando softmax---
    softmax_cpu(host_H, C, n_heads);

    double *device_copy_H = (double *) malloc(n_heads * C * C * sizeof(double));
    hipMemcpy(device_copy_H, device_H, n_heads * C * C * sizeof(double), hipMemcpyDeviceToHost);

    for (int h = 0; h < n_heads; ++h) {
      for (int i = 0; i < C; ++i) {
        for (int j = 0; j < C; ++j) {
          if (fabs(host_H[(h * C * C) + (i * C) + j] - device_copy_H[(h * C * C) + (i * C) + j]) > 1e-9) {
            printf("Deu errado\n");
            printf("head: %d\n", h);
            printf("i: %d j: %d\n", i, j);
            printf("Valor na CPU: %lf\n", host_H[(h * C * C) + (i * C) + j]);
            printf("Valor na GPU: %lf\n", device_copy_H[(h * C * C) + (i * C) + j]);
            return;
          }
        }
      }
    }
    printf("Nada de errado no produto interno escalado!\n");
    printf("Matrizes H:\n");
    for (int h = 0; h < n_heads; ++h) {
      printf("head: %d\n", h);
      for (int i = 0; i < C; ++i) {
        for (int j = 0; j < C; ++j){
          printf("%lf ", host_H[(h * C * C) + (i * C) + j]);
        }
        printf("\n");
      }
    }
    // --- Fim do teste ---
  }
};


int main(){
  srand(998244353);

  double *E = (double *) malloc(C * d_model * sizeof(double));
  for (int i = 0; i < C * d_model; ++i) {
    E[i] = rand_double();
  }
  
  MultiHeadAttention Attention;
  Attention.pass_embedding(E);
}
